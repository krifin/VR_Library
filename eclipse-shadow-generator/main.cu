#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////
//                               This file is part of CosmoScout VR                               //
////////////////////////////////////////////////////////////////////////////////////////////////////

// SPDX-FileCopyrightText: German Aerospace Center (DLR) <cosmoscout@dlr.de>
// SPDX-License-Identifier: MIT

#include "../../src/cs-utils/CommandLine.hpp"

#include "LimbDarkening.cuh"
#include "math.cuh"

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include <stb_image.h>
#include <stb_image_write.h>

////////////////////////////////////////////////////////////////////////////////////////////////////
// This tool can be used to create the eclipse shadow maps used by CosmoScout VR. See the         //
// README.md file in this directory for usage instructions!                                       //
////////////////////////////////////////////////////////////////////////////////////////////////////

// This macro is used in multiple locations to check for Cuda errors.
// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define gpuErrchk(ans)                                                                             \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) {
      exit(code);
    }
  }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

// This is used to pass the command line options to the Cuda kernel.
struct ShadowSettings {
  uint32_t size            = 512;
  bool     includeUmbra    = false;
  double   mappingExponent = 1.0;
};

////////////////////////////////////////////////////////////////////////////////////////////////////

__constant__ LimbDarkening  cLimbDarkening;
__constant__ ShadowSettings cShadowSettings;

////////////////////////////////////////////////////////////////////////////////////////////////////

// Computes the shadow map by sampling the intersection area between circles representing the Sun
// and the occluder. This makes use of the global limb darkening function.
__global__ void computeLimbDarkeningShadow(float* shadowMap) {
  uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
  uint32_t i = y * cShadowSettings.size + x;

  if ((x >= cShadowSettings.size) || (y >= cShadowSettings.size)) {
    return;
  }

  auto angles = math::mapPixelToAngles(glm::ivec2(x, y), cShadowSettings.size,
      cShadowSettings.mappingExponent, cShadowSettings.includeUmbra);

  double sunArea = math::getCircleArea(1.0);

  shadowMap[i] = static_cast<float>(
      1 - math::sampleCircleIntersection(1.0, angles.x, angles.y, cLimbDarkening) / sunArea);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

// Computes the shadow map by analytically computing the intersection area between circles
// representing the Sun and the occluder. This does not use a limb darkening function.
__global__ void computeCircleIntersectionShadow(float* shadowMap) {
  uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
  uint32_t i = y * cShadowSettings.size + x;

  if ((x >= cShadowSettings.size) || (y >= cShadowSettings.size)) {
    return;
  }

  auto angles = math::mapPixelToAngles(glm::ivec2(x, y), cShadowSettings.size,
      cShadowSettings.mappingExponent, cShadowSettings.includeUmbra);

  double sunArea = math::getCircleArea(1.0);

  shadowMap[i] =
      static_cast<float>(1.0 - math::getCircleIntersection(1.0, angles.x, angles.y) / sunArea);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

// Computes the shadow map by assuming a linear brightness gradient from the outer edge of the
// penumbra to the start of the umbra / antumbra. In the antumbra, the shadow intensity decreases
// quadratically. This does not use a limb darkening function.
__global__ void computeLinearShadow(float* shadowMap) {
  uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
  uint32_t i = y * cShadowSettings.size + x;

  if ((x >= cShadowSettings.size) || (y >= cShadowSettings.size)) {
    return;
  }

  auto angles = math::mapPixelToAngles(glm::ivec2(x, y), cShadowSettings.size,
      cShadowSettings.mappingExponent, cShadowSettings.includeUmbra);

  double phiSun = 1.0;
  double phiOcc = angles[0];
  double delta  = angles[1];

  double visiblePortion =
      (delta - glm::abs(phiSun - phiOcc)) / (phiSun + phiOcc - glm::abs(phiSun - phiOcc));

  double maxDepth = glm::min(1.0, glm::pow(phiOcc / phiSun, 2.0));

  shadowMap[i] = static_cast<float>(1.0 - maxDepth * glm::clamp(1.0 - visiblePortion, 0.0, 1.0));
}

////////////////////////////////////////////////////////////////////////////////////////////////////

// Computes the shadow map by assuming a smoothstep-based brightness gradient from the outer edge of
// the penumbra to the start of the umbra / antumbra. In the antumbra, the shadow intensity
// decreases quadratically. This does not use a limb darkening function.
__global__ void computeSmoothstepShadow(float* shadowMap) {
  uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
  uint32_t i = y * cShadowSettings.size + x;

  if ((x >= cShadowSettings.size) || (y >= cShadowSettings.size)) {
    return;
  }

  auto angles = math::mapPixelToAngles(glm::ivec2(x, y), cShadowSettings.size,
      cShadowSettings.mappingExponent, cShadowSettings.includeUmbra);

  double phiSun = 1.0;
  double phiOcc = angles[0];
  double delta  = angles[1];

  double visiblePortion =
      (delta - glm::abs(phiSun - phiOcc)) / (phiSun + phiOcc - glm::abs(phiSun - phiOcc));

  double maxDepth = glm::min(1.0, glm::pow(phiOcc / phiSun, 2.0));

  shadowMap[i] = static_cast<float>(
      1.0 - maxDepth * glm::clamp(1.0 - glm::smoothstep(0.0, 1.0, visiblePortion), 0.0, 1.0));
}

////////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char** argv) {

  stbi_flip_vertically_on_write(1);

  ShadowSettings settings;

  std::string cOutput    = "shadow.hdr";
  std::string cMode      = "limb-darkening";
  bool        cPrintHelp = false;

  // First configure all possible command line options.
  cs::utils::CommandLine args(
      "Welcome to the shadow map generator! Here are the available options:");
  args.addArgument({"-o", "--output"}, &cOutput,
      "The image will be written to this file (default: \"" + cOutput + "\").");
  args.addArgument({"--size"}, &settings.size,
      "The output texture size (default: " + std::to_string(settings.size) + ").");
  args.addArgument({"--mode"}, &cMode,
      "This should be either 'limb-darkening', 'circles', 'linear', or 'smoothstep' (default: " +
          cMode + ").");
  args.addArgument({"--with-umbra"}, &settings.includeUmbra,
      "Add the umbra region to the shadow map (default: " + std::to_string(settings.includeUmbra) +
          ").");
  args.addArgument({"--mapping-exponent"}, &settings.mappingExponent,
      "Adjusts the distribution of sampling positions. A value of 1.0 will position the "
      "umbra's end in the middle of the texture, larger values will shift this to the "
      "right. (default: " +
          std::to_string(settings.mappingExponent) + ").");
  args.addArgument({"-h", "--help"}, &cPrintHelp, "Show this help message.");

  // Then do the actual parsing.
  try {
    std::vector<std::string> arguments(argv + 1, argv + argc);
    args.parse(arguments);
  } catch (std::runtime_error const& e) {
    std::cerr << "Failed to parse command line arguments: " << e.what() << std::endl;
    return 1;
  }

  // When cPrintHelp was set to true, we print a help message and exit.
  if (cPrintHelp) {
    args.printHelp();
    return 0;
  }

  // Check whether a valid mode was given.
  if (cMode != "limb-darkening" && cMode != "circles" && cMode != "linear" &&
      cMode != "smoothstep") {
    std::cerr << "Invalid value given for --mode!" << std::endl;

    return 1;
  }

  // Initialize the limb darkening model.
  LimbDarkening limbDarkening;
  limbDarkening.init();

  // Initialize the global Cuda symbols.
  hipMemcpyToSymbol(HIP_SYMBOL(cShadowSettings), &settings, sizeof(ShadowSettings));
  hipMemcpyToSymbol(HIP_SYMBOL(cLimbDarkening), &limbDarkening, sizeof(LimbDarkening));

  // Compute the 2D kernel size.
  dim3     blockSize(16, 16);
  uint32_t numBlocksX = (settings.size + blockSize.x - 1) / blockSize.x;
  uint32_t numBlocksY = (settings.size + blockSize.y - 1) / blockSize.y;
  dim3     gridSize   = dim3(numBlocksX, numBlocksY);

  // Allocate the shared memory for the shadow map.
  float* shadow = nullptr;
  gpuErrchk(hipMallocManaged(
      &shadow, static_cast<size_t>(settings.size * settings.size) * sizeof(float)));

  // Compute the shadow map based on the given mode.
  if (cMode == "limb-darkening") {
    computeLimbDarkeningShadow<<<gridSize, blockSize>>>(shadow);
  } else if (cMode == "circles") {
    computeCircleIntersectionShadow<<<gridSize, blockSize>>>(shadow);
  } else if (cMode == "linear") {
    computeLinearShadow<<<gridSize, blockSize>>>(shadow);
  } else if (cMode == "smoothstep") {
    computeSmoothstepShadow<<<gridSize, blockSize>>>(shadow);
  }

  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  // Finally write the output texture!
  stbi_write_hdr(
      cOutput.c_str(), static_cast<int>(settings.size), static_cast<int>(settings.size), 1, shadow);

  return 0;
}
